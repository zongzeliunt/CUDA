#include "hip/hip_runtime.h"
//来自chapter05 add_loop_long_blocks.cu

#include "../common/book.h"


/*
__global__ void add( int *a, int *b, int *c ) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < N) {
        c[tid] = a[tid] + b[tid];
        tid += blockDim.x * gridDim.x;
    }
}
*/
//#define N 10
#define N (33 * 1024)

__global__ void add (int *a, int *b, int *c) {
	//int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	printf ("blockdim: %d\n", blockDim.x);	
	printf ("blockIdx: %d\n", blockIdx.x);	
	printf ("gridDim: %d\n", gridDim.x);	
    while (tid < N) {
        c[tid] = a[tid] + b[tid];
        tid += blockDim.x * gridDim.x;
    }
	//c[tid] = a[tid] + b[tid];
	
}

int main (void) {
	int *a, *b, *c;
	int *dev_a, *dev_b, *dev_c;

	a = (int*)malloc(N * sizeof(int));
	b = (int*)malloc(N * sizeof(int));
	c = (int*)malloc(N * sizeof(int));
	
    HANDLE_ERROR( hipMalloc( (void**)&dev_a, N * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_b, N * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_c, N * sizeof(int) ) );

    for (int i=0; i<N; i++) {
        a[i] = i;
        b[i] = 2 * i;
    }

	
    HANDLE_ERROR( hipMemcpy( dev_a, a, N * sizeof(int), hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( dev_b, b, N * sizeof(int), hipMemcpyHostToDevice ) );

	//add<<<N/2, 2>>> (dev_a, dev_b, dev_c);
	add<<<128, 128>>> (dev_a, dev_b, dev_c);

    HANDLE_ERROR( hipMemcpy( c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost ) );

	
    for (int i=0; i<N; i++) {
    	if (c[i] != i + 2 * i) {
			printf ("error: %d", i);
		}
	}
		
    HANDLE_ERROR( hipFree( dev_a ) );
    HANDLE_ERROR( hipFree( dev_b ) );
    HANDLE_ERROR( hipFree( dev_c ) );

    free( a );
    free( b );
    free( c );

	printf ("correct!\n");
    return 0;
}
